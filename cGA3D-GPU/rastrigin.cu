#include <stdio.h>

#include <cstdlib>
#include <iostream>
#include <string>
#include <cstdint>
#include <cstring>
#include <vector>
#include <cassert>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define assertm(exp, msg) assert(((void)msg, exp))
const int C_SIZE  = 32;
const int POINT_POS = 4;


// inds 64
int N_COLS  = 2;
int N_ROWS  = 2;
int N_DEPTH = 2;

int GENS  = 20000;
int dumpLastGeneration = false;

bool is2D = false;

  void initializePopulation(int* chromosome, int offset){
      for(int i=0; i<C_SIZE; i++){
        double randVal = (float) rand()/RAND_MAX ;
        chromosome[offset+i] = 0;
        if (randVal > 0.5)
          chromosome[offset+i] = 1;
      }
  }

  void decodeFixedPointCalculationHost(int* chromosome,int point,  int offset, float* val1, float* val2){
    float x1 = 0;
    float x2 = 0;
    for(int i=1; i<16;i++){
      if (i<point)
      	x1 += pow(2,i-1) * chromosome[offset+i];
      else
      	x1 += pow(2,(i-point)*(-1)) * chromosome[offset+i];
    }

    if (chromosome[offset]==1)  
      x1 = x1 * -1;
    *val1 = x1;

    for(int i=1; i<16;i++){
      if (i<point){
      	x2 += pow(2,i-1) * chromosome[offset+i+16];
      }
      else
      	x2 += pow(2,(i-point)*(-1)) * chromosome[offset+i+16];  
    }                                                     
    if (chromosome[offset+16]==1)
      x2 = x2 * -1;
    *val2 = x2;
  }

  __device__ void decodeFixedPointCalculation(int* chromosome,int point,  int offset, float* val1, float* val2){
    float x1 = 0;
    float x2 = 0;
    for(int i=1; i<16;i++){
      if (i<point)
      	x1 += pow(2,i-1) * chromosome[offset+i];
      else
      	x1 += pow(2,(i-point)*(-1)) * chromosome[offset+i];
    }
    if (chromosome[offset]==1)  
      x1 = x1 * -1;

    for(int i=1; i<16;i++){
      if (i<point){
      	x2 += pow(2,i-1) * chromosome[offset+i+16];
      }
      else
      	x2 += pow(2,(i-point)*(-1)) * chromosome[offset+i+16];  
    }                                                     
    if (chromosome[offset+16]==1)
      x2 = x2 * -1;
    *val1 = x1;
    *val2 = x2;
  }

  __device__ float evaluate(int* chromosome,int offset){
   float x1,x2;
   decodeFixedPointCalculation(chromosome, POINT_POS, offset, &x1, &x2);
   return 20+ x1*x1-cos(2*3.14*x1)  + x2*x2-cos(2*3.14*x2);
  }

__device__ int generateSelection(hiprandState* globalState, int ind)
{
    hiprandState localState = globalState[ind];
    int RANDOM = floor(10*hiprand_uniform( &localState));
    globalState[ind] = localState;
    return RANDOM;
}

__device__ float generateMutation(hiprandState* globalState, int ind)
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed,int N_ROWS, int N_COLS )
{
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  int offset =  i*N_ROWS*N_COLS + j*N_COLS + k;
  hiprand_init( seed, offset , 0, &state[offset]  );
}

int dk_mod(int a,int b) {
  int c = a % b;
  return (c < 0) ? c + b : c;
}


__device__ int kernel_dk_mod(int a,int b) {
  int c = a % b;
  return (c < 0) ? c + b : c;
}


__global__ void KernelTest(float *pop,float *temp_pop, hiprandState* globalState,int N_ROWS, int N_COLS) {
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  int offset = i*N_ROWS*N_COLS + j*N_COLS + k;
  float val = generateSelection(globalState,offset);
  *(temp_pop+offset) = val;
  //temp_pop[offset] = 5;
  printf("Hello, world from the device from thread (%d,%d,%d) offset=%d value=%f!\n",i,j,k,offset,val); 

}

__global__ void KernelCGA2D(int* pop, float* fitness, int* temp_pop, float* temp_fitness, hiprandState* globalState,int N_ROWS, int N_COLS) {
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  
  int offset_chromosome = (i*N_COLS + j)*C_SIZE;
  int offset_fitness    = i*N_COLS + j;
  int offset_rng    = i*N_COLS + j;

  int s = generateSelection(globalState, offset_rng);
  int selectedOffset_fitness    = offset_fitness; // offset selected
  int selectedOffset_chromosome = offset_chromosome; // offset selected
  // performing the selection
  if (s==0){
    selectedOffset_fitness = kernel_dk_mod((i-1),N_ROWS) *N_COLS + j;
  }
  else if (s==1){
    selectedOffset_fitness = kernel_dk_mod((i+1),N_ROWS)*N_COLS + j;
  }
  else if (s==2){
    selectedOffset_fitness = i*N_COLS+ kernel_dk_mod((j-1),N_COLS);
  }
  else if (s>=3){
    selectedOffset_fitness = i*N_COLS+ kernel_dk_mod((j+1),N_COLS);
  }
  selectedOffset_chromosome = selectedOffset_fitness * C_SIZE;

  int child_sequence1[C_SIZE];
  int child_sequence2[C_SIZE];
  int crosspoint = C_SIZE/2;
  // Performing the crossver operation
  for (int p=0; p < C_SIZE; p++){
    if (p < crosspoint){
      child_sequence1[p] = pop[selectedOffset_chromosome+p];
      child_sequence2[p] = pop[offset_chromosome+p];
    }
    else{
      child_sequence1[p] = pop[offset_chromosome+p];
      child_sequence2[p] = pop[selectedOffset_chromosome+p];
    }                                            
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05)    // 5% chance of a random mutation
      if (child_sequence1[p] == 1)
        child_sequence1[p] =0;
      else 
        child_sequence1[p] = 1;
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05) // 5% chance of a random mutation
      if (child_sequence2[p] == 1)
        child_sequence2[p] =0;
      else 
        child_sequence2[p] = 1;
  }
  float fitness_current = evaluate(pop,offset_chromosome);
  float fitness_i1      = evaluate(child_sequence1,0);
  float fitness_i2      = evaluate(child_sequence2,0);
  // decide the best individual
  if (fitness_current >= fitness_i1 && fitness_current  >= fitness_i2){
    // copy current in temp
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = pop[offset_chromosome+c];
    }	  
    temp_fitness[offset_fitness]  = fitness[offset_fitness];
  }
  else if(fitness_i1 >= fitness_current && fitness_i1 >= fitness_i2){
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
  else{
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
}

__global__ void KernelCGA(int* pop, float* fitness, int* temp_pop, float* temp_fitness, hiprandState* globalState,int N_ROWS, int N_COLS) {
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  
  int offset_chromosome = (i*N_ROWS*N_COLS + j*N_COLS + k)*C_SIZE;
  int offset_fitness    = i*N_ROWS*N_COLS + j*N_COLS + k;
  int offset_rng    = i*N_ROWS*N_COLS + j*N_COLS + k;

  int s = generateSelection(globalState, offset_rng);
  int selectedOffset_fitness    = offset_fitness; // offset selected
  int selectedOffset_chromosome = offset_chromosome; // offset selected
  // performing the selection
  if (s==0){
    selectedOffset_fitness = kernel_dk_mod((i-1),N_ROWS)*N_ROWS *N_COLS + j*N_COLS + k;
  }
  else if (s==1){
    selectedOffset_fitness = kernel_dk_mod((i+1),N_ROWS)*N_ROWS *N_COLS + j*N_COLS + k;
  }
  else if (s==2){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ kernel_dk_mod((j-1),N_COLS)*N_COLS+k;
  }
  else if (s==3){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ kernel_dk_mod((j+1),N_COLS)*N_COLS+k;
  }
  else if (s==4){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ j*N_COLS+ kernel_dk_mod((k+1),N_COLS) ;
  }
  else if (s>5){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ j*N_COLS+ kernel_dk_mod((k-1),N_COLS) ;
  }
  selectedOffset_chromosome = selectedOffset_fitness * C_SIZE;

  int child_sequence1[C_SIZE];
  int child_sequence2[C_SIZE];
  int crosspoint = C_SIZE/2;
  // Performing the crossver operation
  for (int p=0; p < C_SIZE; p++){
    if (p < crosspoint){
      child_sequence1[p] = pop[selectedOffset_chromosome+p];
      child_sequence2[p] = pop[offset_chromosome+p];
    }
    else{
      child_sequence1[p] = pop[offset_chromosome+p];
      child_sequence2[p] = pop[selectedOffset_chromosome+p];
    }                                            
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05)    // 5% chance of a random mutation
      if (child_sequence1[p] == 1)
        child_sequence1[p] =0;
      else 
        child_sequence1[p] = 1;
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05) // 5% chance of a random mutation
      if (child_sequence2[p] == 1)
        child_sequence2[p] =0;
      else 
        child_sequence2[p] = 1;
  }
  float fitness_current = evaluate(pop,offset_chromosome);
  float fitness_i1      = evaluate(child_sequence1,0);
  float fitness_i2      = evaluate(child_sequence2,0);
  // decide the best individual
  if (fitness_current >= fitness_i1 && fitness_current  >= fitness_i2){
    // copy current in temp
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = pop[offset_chromosome+c];
    }	  
    temp_fitness[offset_fitness]  = fitness[offset_fitness];
  }
  else if(fitness_i1 >= fitness_current && fitness_i1 >= fitness_i2){
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
  else{
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
}

void cGACuda(){
  hiprandState* devStates;
  hipMalloc (&devStates, N_ROWS * N_COLS * N_DEPTH * sizeof(hiprandState));
  srand(time(0));
  /** ADD THESE TWO LINES **/
  int seed = rand();
  int numBlocks = 1;
  dim3 threadsPerBlock(N_ROWS, N_COLS, N_DEPTH);
  setup_kernel<<<numBlocks,threadsPerBlock>>>(devStates,seed, N_ROWS, N_COLS);
  int *pop = (int*)malloc(N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int));       // array on the host machine
  float *fitness = (float*)malloc(N_ROWS * N_COLS * N_DEPTH * sizeof(float));          // array on the host machinei
  int *pop_gpu,*tem_pop_gpu;   // arrays in the gpu memory
  float *fitness_gpu,*tem_fitness_gpu;   // arrays in the gpu memory

  if (pop == NULL)
  {
      fprintf(stderr, "Out of memory");
      exit(0);
  }
  // randomly initializing the population
  //std::cout << "Init Memory " << std::endl;
  for(int i=0;i < N_ROWS;i++ ){       
    for(int j=0; j < N_COLS; j++){
      for(int k=0; k < N_DEPTH; k++){
        for(int c=0; c< C_SIZE; c++){
	  initializePopulation(pop,(i*N_ROWS*N_COLS + j*N_COLS + k) * C_SIZE);;
	}
	fitness[ i*N_ROWS*N_COLS + j*N_COLS + k] = 0.0;
      }
    }
  }
  // 2.c. allocate the memory on the GPU
  hipMalloc( (void**)&pop_gpu, N_ROWS * N_COLS * N_DEPTH* C_SIZE * sizeof(int));
  hipMalloc( (void**)&tem_pop_gpu, N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int));
  hipMalloc( (void**)&fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(float));
  hipMalloc( (void**)&tem_fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(float));

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  for(int x=0;x<GENS;x++){
    //std::cout << "It:" << x << std::endl;
    // 2.d. copy the arrays 'a' and 'b' to the GPU
   hipMemcpy(pop_gpu     , pop     , N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int) , hipMemcpyHostToDevice );
   hipMemcpy(fitness_gpu , fitness , N_ROWS * N_COLS * N_DEPTH * sizeof(float) , hipMemcpyHostToDevice );
  // here calls the kernel to be executed on the gpu
  KernelCGA<<<numBlocks,threadsPerBlock >>>(pop_gpu,fitness_gpu,tem_pop_gpu,tem_fitness_gpu,devStates, N_ROWS, N_COLS);
  //cudaDeviceSynchronize();
  //std::cout << "Error: " << cudaGetErrorString(cudaGetLastError()) << '\n';
  // here read back the result from the kernel
  hipMemcpy( pop, tem_pop_gpu, N_ROWS * N_COLS * N_DEPTH * C_SIZE*sizeof(int)  , hipMemcpyDeviceToHost );
  hipMemcpy( fitness, tem_fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(float)  , hipMemcpyDeviceToHost );
  }
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout <<  std::chrono::duration_cast<std::chrono::milliseconds> (end - begin).count() << "[ms]" << std::endl;
  if(dumpLastGeneration){
    std::cout << "Dumping result " << std::endl;
    for(int i=0;i < N_ROWS;i++ ){       
      for(int j=0; j < N_COLS; j++){
        for(int k=0; k < N_DEPTH; k++){
          std::cout << "C: ";
          for(int c=0; c < C_SIZE; c++){ 
            float x1, x2;
	    decodeFixedPointCalculationHost(pop,POINT_POS,(i*N_ROWS*N_COLS + j*N_COLS + k) * C_SIZE+ c ,&x1,&x2);
            std::cout <<"(" <<x1 << "," << x2 << ")" << ",";
          }
          std::cout <<std::endl << "F:" << fitness[i*N_ROWS*N_COLS + j*N_COLS + k] << std::endl;
        }
      }
    }
  }
  // free the memory allocated in the host
  free(pop);
  free(fitness); 
  // free the memory we allocated on the GPU
  hipFree( pop_gpu );
  hipFree( tem_pop_gpu );
  hipFree( fitness_gpu );
  hipFree( tem_fitness_gpu );
}

void cGACuda2D(){
  hiprandState* devStates;
  hipMalloc (&devStates, N_ROWS * N_COLS * sizeof(hiprandState));
  srand(time(0));
  /** ADD THESE TWO LINES **/
  int seed = rand();
  int numBlocks = 1;
  dim3 threadsPerBlock(N_ROWS, N_COLS);
  setup_kernel<<<numBlocks,threadsPerBlock>>>(devStates,seed, N_ROWS, N_COLS);
  int *pop = (int*)malloc(N_ROWS * N_COLS *  C_SIZE* sizeof(int));       // array on the host machine
  float *fitness = (float*)malloc(N_ROWS * N_COLS *  sizeof(float));          // array on the host machinei
  int *pop_gpu,*tem_pop_gpu;   // arrays in the gpu memory
  float *fitness_gpu,*tem_fitness_gpu;   // arrays in the gpu memory

  if (pop == NULL)
  {
      fprintf(stderr, "Out of memory");
      exit(0);
  }
  // randomly initializing the population
  //std::cout << "Init Memory " << std::endl;
  for(int i=0;i < N_ROWS;i++ ){       
    for(int j=0; j < N_COLS; j++){
      for(int c=0; c< C_SIZE; c++){
        initializePopulation(pop,(i*N_COLS + j) * C_SIZE);;
      }
      fitness[ i*N_COLS + j] = 0,0;
    }
  }
  // 2.c. allocate the memory on the GPU
  hipMalloc( (void**)&pop_gpu, N_ROWS * N_COLS* C_SIZE * sizeof(int));
  hipMalloc( (void**)&tem_pop_gpu, N_ROWS * N_COLS * C_SIZE* sizeof(int));
  hipMalloc( (void**)&fitness_gpu, N_ROWS * N_COLS * sizeof(float));
  hipMalloc( (void**)&tem_fitness_gpu, N_ROWS * N_COLS * sizeof(float));

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  for(int x=0;x<GENS;x++){
    //std::cout << "It:" << x << std::endl;
    // 2.d. copy the arrays 'a' and 'b' to the GPU
   hipMemcpy(pop_gpu     , pop     , N_ROWS * N_COLS  * C_SIZE* sizeof(int) , hipMemcpyHostToDevice );
   hipMemcpy(fitness_gpu , fitness , N_ROWS * N_COLS  * sizeof(float) , hipMemcpyHostToDevice );
  // here calls the kernel to be executed on the gpu
  KernelCGA2D<<<numBlocks,threadsPerBlock >>>(pop_gpu,fitness_gpu,tem_pop_gpu,tem_fitness_gpu,devStates, N_ROWS, N_COLS);
  //cudaDeviceSynchronize();
  //std::cout << "Error: " << cudaGetErrorString(cudaGetLastError()) << '\n';
  // here read back the result from the kernel
  hipMemcpy( pop, tem_pop_gpu, N_ROWS * N_COLS * C_SIZE*sizeof(int)  , hipMemcpyDeviceToHost );
  hipMemcpy( fitness, tem_fitness_gpu, N_ROWS * N_COLS * sizeof(float)  , hipMemcpyDeviceToHost );
  }
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout <<  std::chrono::duration_cast<std::chrono::milliseconds> (end - begin).count() << "[ms]" << std::endl;
  if(dumpLastGeneration){
    std::cout << "Dumping result " << std::endl;
    for(int i=0;i < N_ROWS;i++ ){       
      for(int j=0; j < N_COLS; j++){
        std::cout << "C: ";
        for(int c=0; c < C_SIZE; c++){ 
	  float x1, x2;
	  decodeFixedPointCalculationHost(pop,POINT_POS,(i*N_COLS + j) * C_SIZE+ c ,&x1,&x2);
	  std::cout <<"(" <<x1 << "," << x2 << ")" << ",";
        }
        std::cout <<std::endl << "F:" << fitness[i*N_COLS + j] << std::endl;
      }
    }
  }
  // free the memory allocated in the host
  free(pop);
  free(fitness); 
  // free the memory we allocated on the GPU
  hipFree( pop_gpu );
  hipFree( tem_pop_gpu );
  hipFree( fitness_gpu );
  hipFree( tem_fitness_gpu );
}

int main() {
  if(const char* env_p = std::getenv("GENS")){
        std::string s = env_p;
        GENS = stoi(s);
  }
  if(const char* env_p = std::getenv("COLS")){
        std::string s = env_p;
        N_COLS = stoi(s);
  } 
  if(const char* env_p = std::getenv("ROWS")){
        std::string s = env_p;
        N_ROWS = stoi(s);
  } 
  if(const char* env_p = std::getenv("DEPTH")){
        std::string s = env_p;
        N_DEPTH = stoi(s);
  } 
  else
	is2D = true;
  if(const char* env_p = std::getenv("DUMP")){
        //std::string s = env_p;
        //N_DEPTH = stoi(s);
	 dumpLastGeneration = true;
  }
  int exps = 1;
  if(const char* env_p = std::getenv("EXP")){
        std::string s = env_p;
        exps = stoi(s);
  }
  for(int i=0; i< exps; i++)
    if (is2D){	  
      //std::cout << "is2D" << std::endl;
      cGACuda2D();
    }
    else	    
     cGACuda();

  return 0;
}


