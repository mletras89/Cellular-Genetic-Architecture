#include <stdio.h>

#include <cstdlib>
#include <iostream>
#include <string>
#include <cstdint>
#include <cstring>
#include <vector>
#include <cassert>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Use (void) to silence unused warnings.
#define assertm(exp, msg) assert(((void)msg, exp))
const int C_SIZE  = 32;

// inds 64
int N_COLS  = 2;
int N_ROWS  = 2;
int N_DEPTH = 2;

//const int N_COLS  = 4;
//const int N_ROWS  = 4;
//const int N_DEPTH = 4;

//const int N_COLS  = 8;
//const int N_ROWS  = 8;
//const int N_DEPTH = 1;

int GENS  = 20000;


  void initializePopulation(int* chromosome, int offset){
      for(int i=0; i<C_SIZE; i++){
        double randVal = (float) rand()/RAND_MAX ;
        chromosome[offset+i] = 0;
        if (randVal > 0.5)
          chromosome[offset+i] = 1;
      }
  }

  __device__ int evaluate(int* chromosome,int offset){
// ------------------------------
// X    00    01    10    11
//-------------------------------
//Iso1  m     0     0     m-a
//Iso2  0     0     0     m
//-------------------------------
    int fitness = 0;
    if (chromosome[offset] == 1 && chromosome[offset+1] == 1)
      fitness += 2;
    for(int i=2; i< 16 ;i++){
      if(chromosome[offset+2*i-1]==0 && chromosome[offset+2*i-2]==0)
        fitness+= 2;
      else if(chromosome[offset+2*i-1]==1 && chromosome[offset+2*i-2]==1)
        fitness += 1;
    }
    return fitness;
  }

__device__ int generateSelection(hiprandState* globalState, int ind)
{
    hiprandState localState = globalState[ind];
    int RANDOM = floor(10*hiprand_uniform( &localState));
    globalState[ind] = localState;
    return RANDOM;
}

__device__ float generateMutation(hiprandState* globalState, int ind)
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed,int N_ROWS, int N_COLS )
{
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  int offset =  i*N_ROWS*N_COLS + j*N_COLS + k;
  hiprand_init( seed, offset , 0, &state[offset]  );
}

int dk_mod(int a,int b) {
  int c = a % b;
  return (c < 0) ? c + b : c;
}


__device__ int kernel_dk_mod(int a,int b) {
  int c = a % b;
  return (c < 0) ? c + b : c;
}


__global__ void KernelTest(float *pop,float *temp_pop, hiprandState* globalState,int N_ROWS, int N_COLS) {
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  int offset = i*N_ROWS*N_COLS + j*N_COLS + k;
  float val = generateSelection(globalState,offset);
  *(temp_pop+offset) = val;
  //temp_pop[offset] = 5;
  printf("Hello, world from the device from thread (%d,%d,%d) offset=%d value=%f!\n",i,j,k,offset,val); 

}

__global__ void KernelCGA(int* pop, int* fitness, int* temp_pop, int* temp_fitness, hiprandState* globalState,int N_ROWS, int N_COLS) {
  int i = threadIdx.x; 
  int j = threadIdx.y; 
  int k = threadIdx.z;
  
  int offset_chromosome = (i*N_ROWS*N_COLS + j*N_COLS + k)*C_SIZE;
  int offset_fitness    = i*N_ROWS*N_COLS + j*N_COLS + k;
  int offset_rng    = i*N_ROWS*N_COLS + j*N_COLS + k;

  int s = generateSelection(globalState, offset_rng);
  int selectedOffset_fitness    = offset_fitness; // offset selected
  int selectedOffset_chromosome = offset_chromosome; // offset selected

  if (s==0){
    selectedOffset_fitness = kernel_dk_mod((i-1),N_ROWS)*N_ROWS *N_COLS + j*N_COLS + k;
  }
  else if (s==1){
    selectedOffset_fitness = kernel_dk_mod((i+1),N_ROWS)*N_ROWS *N_COLS + j*N_COLS + k;
  }
  else if (s==2){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ kernel_dk_mod((j-1),N_COLS)*N_COLS+k;
  }
  else if (s==3){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ kernel_dk_mod((j+1),N_COLS)*N_COLS+k;
  }
  else if (s==4){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ j*N_COLS+ kernel_dk_mod((k+1),N_COLS) ;
  }
  else if (s>5){
    selectedOffset_fitness = i*N_ROWS*N_COLS+ j*N_COLS+ kernel_dk_mod((k-1),N_COLS) ;
  }
  selectedOffset_chromosome = selectedOffset_fitness * C_SIZE;

  int child_sequence1[C_SIZE];
  int child_sequence2[C_SIZE];
  int crosspoint = C_SIZE/2;
  // Performing the crossver operation
  for (int p=0; p < C_SIZE; p++){
    if (p < crosspoint){
      child_sequence1[p] = pop[selectedOffset_chromosome+p];
      child_sequence2[p] = pop[offset_chromosome+p];
    }
    else{
      child_sequence1[p] = pop[offset_chromosome+p];
      child_sequence2[p] = pop[selectedOffset_chromosome+p];
    }                                            
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05)    // 5% chance of a random mutation
      if (child_sequence1[p] == 1)
        child_sequence1[p] =0;
      else 
        child_sequence1[p] = 1;
  }
  // perform the mutation
  for (int p=0; p<C_SIZE; p++){
    double randVal = generateMutation(globalState,offset_rng);
    if (randVal < 0.05) // 5% chance of a random mutation
      if (child_sequence2[p] == 1)
        child_sequence2[p] =0;
      else 
        child_sequence2[p] = 1;
  }
  int fitness_current = evaluate(pop,offset_chromosome);
  int fitness_i1      = evaluate(child_sequence1,0);
  int fitness_i2      = evaluate(child_sequence2,0);
  // decide the best individual
  if (fitness_current >= fitness_i1 && fitness_current  >= fitness_i2){
    // copy current in temp
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = pop[offset_chromosome+c];
    }	  
    temp_fitness[offset_fitness]  = fitness[offset_fitness];
  }
  else if(fitness_i1 >= fitness_current && fitness_i1 >= fitness_i2){
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
  else{
    // copy current in ind1
    for (int c=0; c<C_SIZE; c++){ 
      temp_pop[offset_chromosome+c] = child_sequence1[c];
    }	  
    temp_fitness[offset_fitness]  = fitness_i1;
  }
//  for(int c=0 ; c<2; c++){
//    temp_pop[offset_chromosome+c] = 1;
//  }
//  temp_fitness[offset_fitness] = evaluate(temp_pop,offset_chromosome);
//
//  printf("Device from thread (%d,%d,%d) offsetPop=%d offsetF=%d!\n",i,j,k,offset_chromosome,offset_fitness); 
}

void cGACuda(){
  hiprandState* devStates;
  hipMalloc (&devStates, N_ROWS * N_COLS * N_DEPTH * sizeof(hiprandState));
  srand(time(0));
  /** ADD THESE TWO LINES **/
  int seed = rand();
  int numBlocks = 1;
  dim3 threadsPerBlock(N_ROWS, N_COLS, N_DEPTH);
  setup_kernel<<<numBlocks,threadsPerBlock>>>(devStates,seed, N_ROWS, N_COLS);

  int *pop = (int*)malloc(N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int));       // array on the host machine
  int *fitness = (int*)malloc(N_ROWS * N_COLS * N_DEPTH * sizeof(int));          // array on the host machine
  int *pop_gpu,*tem_pop_gpu;   // arrays in the gpu memory
  int *fitness_gpu,*tem_fitness_gpu;   // arrays in the gpu memory


  if (pop == NULL)
  {
      fprintf(stderr, "Out of memory");
      exit(0);
  }
// init pop
  std::cout << "Init Memory " << std::endl;
  for(int i=0;i < N_ROWS;i++ ){       
    for(int j=0; j < N_COLS; j++){
      for(int k=0; k < N_DEPTH; k++){
        for(int c=0; c< C_SIZE; c++){
	  initializePopulation(pop,(i*N_ROWS*N_COLS + j*N_COLS + k) * C_SIZE);;
	}
	fitness[ i*N_ROWS*N_COLS + j*N_COLS + k] = 5;
      }
    }
  }
  // 2.c. allocate the memory on the GPU
  hipMalloc( (void**)&pop_gpu, N_ROWS * N_COLS * N_DEPTH* C_SIZE * sizeof(int));
  hipMalloc( (void**)&tem_pop_gpu, N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int));
  hipMalloc( (void**)&fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(int));
  hipMalloc( (void**)&tem_fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(int));

  for(int x=0;x<GENS;x++){
    std::cout << "It:" << x << std::endl;
    // 2.d. copy the arrays 'a' and 'b' to the GPU
   hipMemcpy(pop_gpu     , pop     , N_ROWS * N_COLS * N_DEPTH * C_SIZE* sizeof(int) , hipMemcpyHostToDevice );
   hipMemcpy(fitness_gpu , fitness , N_ROWS * N_COLS * N_DEPTH * sizeof(int) , hipMemcpyHostToDevice );

  // here calls the kernel
  KernelCGA<<<numBlocks,threadsPerBlock >>>(pop_gpu,fitness_gpu,tem_pop_gpu,tem_fitness_gpu,devStates, N_ROWS, N_COLS);
  hipDeviceSynchronize();
  std::cout << "Error: " << hipGetErrorString(hipGetLastError()) << '\n';
  //KernelCGA(int pop[][][], int fitness[][][], int temp_pop[][][], int temp_fitness, curandState* globalState)

  // here read back the result from the kernel
  hipMemcpy( pop, tem_pop_gpu, N_ROWS * N_COLS * N_DEPTH * C_SIZE*sizeof(int)  , hipMemcpyDeviceToHost );
  hipMemcpy( fitness, tem_fitness_gpu, N_ROWS * N_COLS * N_DEPTH * sizeof(int)  , hipMemcpyDeviceToHost );
  }
  std::cout << "Dumping result " << std::endl;
  for(int i=0;i < N_ROWS;i++ ){       
    for(int j=0; j < N_COLS; j++){
      for(int k=0; k < N_DEPTH; k++){
	std::cout << "C: ";
	for(int c=0; c < C_SIZE; c++){ 
	  std::cout << pop[(i*N_ROWS*N_COLS + j*N_COLS + k) * C_SIZE+ c] << ",";
	}
	std::cout <<std::endl << "F:" << fitness[i*N_ROWS*N_COLS + j*N_COLS + k] << std::endl;
      }
    }
  }
  free(pop);
  free(fitness); 
  // free the memory we allocated on the GPU
  hipFree( pop_gpu );
  hipFree( tem_pop_gpu );
  hipFree( fitness_gpu );
  hipFree( tem_fitness_gpu );
}

int main() {
  if(const char* env_p = std::getenv("GENS")){
        std::string s = env_p;
        GENS = stoi(s);
  }
  if(const char* env_p = std::getenv("COLS")){
        std::string s = env_p;
        N_COLS = stoi(s);
  } 
  if(const char* env_p = std::getenv("ROWS")){
        std::string s = env_p;
        N_ROWS = stoi(s);
  } 
  if(const char* env_p = std::getenv("DEPTH")){
        std::string s = env_p;
        N_DEPTH = stoi(s);
  } 
  cGACuda();

  return 0;
}


